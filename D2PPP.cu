#include "hip/hip_runtime.h"
// ROOT stuff
#include <TApplication.h>
#include <TCanvas.h>
#include <TFile.h>
#include <TH1F.h>
#include <TH2F.h>
#include <TGraph.h>
#include <TLegend.h>
#include <TMath.h>
#include <TRandom.h>
#include <TRandom3.h>
#include <TTree.h>
#include <TROOT.h>
#include <TMinuit.h>
#include <TNtuple.h>


// System stuff
#include <CLI/Timer.hpp>
#include <fstream>


// GooFit stuff
#include <goofit/Application.h>
#include <goofit/BinnedDataSet.h>
#include <goofit/FitManager.h>
#include <goofit/fitting/FitManagerMinuit2.h>
#include <goofit/PDFs/GooPdf.h>
#include <goofit/PDFs/basic/PolynomialPdf.h>
#include <goofit/PDFs/basic/SmoothHistogramPdf.h>
#include <goofit/PDFs/combine/AddPdf.h>
#include <goofit/PDFs/combine/ProdPdf.h>
#include <goofit/PDFs/physics/DalitzPlotPdf.h>
#include <goofit/PDFs/physics/DalitzVetoPdf.h>
#include <goofit/PDFs/physics/ResonancePdf.h>
#include <goofit/UnbinnedDataSet.h>
#include <goofit/Variable.h>
#include <goofit/PDFs/physics/DalitzPlotter.h>


#include <thrust/transform_reduce.h>

using namespace std;
using namespace GooFit;
using namespace ROOT;


//Globals

double pi_MASS  = 0.13957018; //GEV
double D_MASS   = 1.86959; //GEV

double d1_MASS  = pi_MASS;  //daughter 1 mass
double d2_MASS  = pi_MASS;
double d3_MASS  = pi_MASS;


fptype s12_min = POW2(d1_MASS  + d2_MASS);
fptype s12_max = POW2(D_MASS   - d2_MASS);
fptype s13_min = POW2(d1_MASS  + d3_MASS);
fptype s13_max = POW2(D_MASS   - d2_MASS);

Observable s12("s12",s12_min,s12_max); //s12^{2}
Observable s13("s13",s13_min,s13_max);
EventNumber eventNumber("eventNumber");

DalitzPlotPdf* signaldalitz = nullptr;
UnbinnedDataSet* Data = nullptr;
std::vector<PdfBase *> comps;

Variable massSum("massSum", POW2(D_MASS) + POW2(d1_MASS) + POW2(d2_MASS) + POW2(d3_MASS));

//functions

DalitzPlotPdf *makesignalpdf(GooPdf *eff = 0);

void maketoydalitzdata(GooPdf* overallsignal,std::string name, size_t nEvents){

DalitzPlotter dp(overallsignal,signaldalitz);

Data = new UnbinnedDataSet({s12,s13,eventNumber});

    std::cout << "Toy Generation begin!" << '\n';
    {
        TCanvas foo;
        auto th1 = dp.make2D();
        th1->Rebin2D(5,5);
        th1->GetXaxis()->SetTitle("#pi^{-}#pi^{+} [Gev/c^{2}]");
        th1->GetYaxis()->SetTitle("#pi^{-}#pi^{+} [Gev/c^{2}]");
        th1->SetStats(0);
        th1->Draw("COLZ");
        foo.SaveAs("plots/PDF.png");
        std::cout << "PDF plotted" << '\n';
    }

        dp.fillDataSetMC(*Data,nEvents);
        TH2F th2("toyData", "", 200, s12.getLowerLimit(), s12.getUpperLimit(), 200, s13.getLowerLimit(),
                         s13.getUpperLimit());
    th2.GetXaxis()->SetTitle("#pi^{-}#pi^{+} [Gev/c^{2}]");
    th2.GetYaxis()->SetTitle("#pi^{-}#pi^{+} [Gev/c^{2}]");

    {
        ofstream w(name);

            for (size_t i = 0; i < Data->getNumEvents(); i++) {
                Data->loadEvent(i);
                th2.Fill(s12, s13);
                w << i << "\t" << std::setprecision(6) << s12.getValue() << "\t" << s13.getValue() << '\n';

            }

            std::cout << "nEvents generated = " << Data->getNumEvents() << '\n';

        w.close();

    }

    TCanvas foo;
    th2.Draw("COLZ");
    th2.SetStats(0);
    foo.SaveAs("plots/toyData.png");

    std::cout << "toy data plotted" << '\n';
    std::cout << "toy Generation end!" << '\n';




}

DalitzPlotPdf* makesignalpdf(GooPdf* eff){

    DecayInfo3 dtoppp;
    dtoppp.motherMass   = D_MASS;
    dtoppp.daug1Mass    = d1_MASS;
    dtoppp.daug2Mass    = d2_MASS;
    dtoppp.daug3Mass    = d3_MASS;
    dtoppp.meson_radius = 1.5;

    //Mass and width
    double rho_MASS     = 0.77526;
    double rho_WIDTH    = 0.1478;
    double rho_amp      = 1.0;
    double rho_phase    = 0.0;

    double omega_MASS   = 0.78265;
    double omega_WIDTH  = 0.00849;
    double omega_amp    = 10.0;
    double omega_phase  = 20*M_PI/180.0;

    double f2_MASS     = 1.2755;
    double f2_WIDTH    = 0.1867;
    double f2_amp      = 15.0;
    double f2_phase    = 30*M_PI/180.0;

    //rho(770)
    Variable v_rho_Mass("rho_MASS",rho_MASS,0.00025,rho_MASS*0.95,rho_MASS*1.1);
    Variable v_rho_Width("rho_WIDTH",rho_WIDTH,0.0009,rho_WIDTH*0.95,rho_WIDTH*1.1);
    Variable v_rho_amp_real("rho_amp_real",rho_amp*cos(rho_phase));
    Variable v_rho_amp_img("rho_amp_img",rho_amp*sin(rho_phase));

    v_rho_Mass.setFixed(true);
    v_rho_Width.setFixed(true);

    //omega(782)
    Variable v_omega_Mass("omega_MASS",omega_MASS);
    Variable v_omega_Width("omega_WIDTH",omega_WIDTH);
    Variable v_omega_amp_real("omega_amp_real",omega_amp*cos(omega_phase),0.001, -100, +100);
    Variable v_omega_amp_img("omega_amp_img",omega_amp*sin(omega_phase),0.001, -100, +100);

    //f2(1270)
    Variable v_f2_Mass("f2_MASS",f2_MASS);
    Variable v_f2_Width("f2_WIDTH",f2_WIDTH);
    Variable v_f2_amp_real("f2_amp_real",f2_amp*cos(f2_phase),0.001, -100, +100);
    Variable v_f2_amp_img("f2_amp_img",f2_amp*sin(f2_phase),0.001, -100, +100);

    //NR

    Variable nonr_amp_real("nonr_amp_real", 1.0, 0.001, -100, +100);
    Variable nonr_amp_imag("nonr_amp_imag", 0.0, 0.001, -100, +100);

    //setting resonances
    ResonancePdf* rho_12 = new Resonances::GS("rho",v_rho_amp_real,v_rho_amp_img,v_rho_Mass,v_rho_Width,(unsigned int)0,PAIR_12);
    ResonancePdf* rho_13 = new Resonances::GS("rho",v_rho_amp_real,v_rho_amp_img,v_rho_Mass,v_rho_Width,(unsigned int)0,PAIR_13);

    ResonancePdf* omega_12 = new Resonances::GS("omega",v_omega_amp_real,v_omega_amp_img,v_omega_Mass,v_omega_Width,(unsigned int)0,PAIR_12);
    ResonancePdf* omega_13 = new Resonances::GS("omega",v_omega_amp_real,v_omega_amp_img,v_omega_Mass,v_omega_Width,(unsigned int)0,PAIR_13);

    ResonancePdf* f2 = new Resonances::RBW("f2",v_f2_amp_real,v_f2_amp_img,v_f2_Mass,v_f2_Width,(unsigned int)0,PAIR_12,true);
    ResonancePdf *nonr = new Resonances::NonRes("nonr", nonr_amp_real, nonr_amp_imag);


    dtoppp.resonances.push_back(rho_12);
    dtoppp.resonances.push_back(rho_13);
    dtoppp.resonances.push_back(omega_12);
    dtoppp.resonances.push_back(omega_13);
    //dtoppp.resonances.push_back(f2);
    //dtoppp.resonances.push_back(nonr);


    if(!eff) {
        // By default create a constant efficiency.
        vector<Variable> offsets;
        vector<Observable> observables;
        vector<Variable> coefficients;
        Variable constantOne("constantOne", 1);
        Variable constantZero("constantZero", 0);

        observables.push_back(s12);
        observables.push_back(s13);
        offsets.push_back(constantZero);
        offsets.push_back(constantZero);
        coefficients.push_back(constantOne);
        eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0); //No efficiency
    }

    return new DalitzPlotPdf("signalPDF", s12, s13, eventNumber, dtoppp, eff);
}

void gettoydata(std::string name){

    std::cout << "get data begin!" << '\n';

    Data = new UnbinnedDataSet({s12,s13,eventNumber});

    std::ifstream reader(name.c_str());

    while(reader >> eventNumber >> s12 >> s13){
        Data->addEvent();
    }

    reader.close();
    std::cout << "get data end!" << '\n';
}

void runtoygen(std::string name, size_t events){

    s12.setNumBins(1500);
    s13.setNumBins(1500);

    signaldalitz = makesignalpdf(0);

    std::cout << "Creating Overall PDF" << '\n';
    ProdPdf* overallSignal = new ProdPdf("overallSignal",{signaldalitz});
    {
        maketoydalitzdata(overallSignal,name,events);
    }
}

void PrintFF(std::vector<std::vector<fptype>> ff){

    size_t nEntries = signaldalitz->getCachedWave(0).size();
    size_t n_res = signaldalitz->getDecayInfo().resonances.size();
    fptype sum = 0;

    std::cout << "nEntries= " << nEntries << '\n';
    for(size_t i = 0; i < n_res ; i++){

        for(size_t j = 0; j< n_res ; j++){
            std::cout << "FF[" << i << "," << j <<"]= " << ff[i][j] << std::endl;

        }

        sum+=ff[i][i];
    }

    std::cout << "Sum[i,i]= " << sum << std::endl;
}

void makeplot(ResonancePdf* comp , GooPdf* eff){

    if(signaldalitz== nullptr){
        signaldalitz = makesignalpdf(0);
    }

    DecayInfo3 temp;
    temp.motherMass  =  signaldalitz->getDecayInfo().motherMass;
    temp.daug1Mass   =   signaldalitz->getDecayInfo().daug1Mass;
    temp.daug2Mass   =   signaldalitz->getDecayInfo().daug2Mass;
    temp.daug3Mass   =   signaldalitz->getDecayInfo().daug3Mass;

    temp.resonances.push_back(comp);

    if(!eff) {
        // By default create a constant efficiency.
        vector<Variable> offsets;
        vector<Observable> observables;
        vector<Variable> coefficients;
        Variable constantOne("constantOne", 1);
        Variable constantZero("constantZero", 0);

        observables.push_back(s12);
        observables.push_back(s13);
        offsets.push_back(constantZero);
        offsets.push_back(constantZero);
        coefficients.push_back(constantOne);
        eff = new PolynomialPdf("constantEff", observables, coefficients, offsets, 0); //No efficiency
    }

    DalitzPlotPdf* signal = new DalitzPlotPdf("signal",s12,s13,eventNumber,temp,eff);
    ProdPdf* overallsignal = new ProdPdf("overallsignal",{signal});

    Data = new UnbinnedDataSet({s12,s13,eventNumber});

    DalitzPlotter temp_dp((GooPdf*)overallsignal,signal);
    temp_dp.fillDataSetMC(*Data,10000);

    TH1F hist("hist","",100,s12.getLowerLimit(),s12.getUpperLimit());

    for(size_t i = 0 ; i < Data->getNumEvents(); i++){
        Data->loadEvent(i);
        hist.Fill(s12.getValue());
    }

    TCanvas foo;
    hist.Draw();
    foo.SaveAs("plots/test.png");

}

void drawFitPlotsWithPulls(TH1 *hd, TH1 *ht, string plotdir) {
    const char *hname = hd->GetName();
    char obsname[10];
    for(int i = 0;; i++) {
        if(hname[i] == '_')
            obsname[i] = '\0';
        else
            obsname[i] = hname[i];
        if(obsname[i] == '\0')
            break;
    }
    ht->Scale(hd->Integral() / ht->Integral()*5);
    ht->SetLineColor(kRed);
    ht->SetLineWidth(3);
    ht->SetMarkerStyle(0);

    hd->SetMarkerColor(kBlack);
    hd->Rebin(5);


    TCanvas foo;

    hd->Draw("E");
    ht->Draw("HIST C same");


    foo.SaveAs(TString::Format("plots/%s_fit.png",obsname));


}


void makeToyDalitzPdfPlots(GooPdf *overallSignal, string plotdir = "plots") {
    TH1F s12_dat_hist("s12_dat_hist", "", s12.getNumBins(), s12.getLowerLimit(), s12.getUpperLimit());
    s12_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV]");
    s12_dat_hist.GetYaxis()->SetTitle(TString::Format("Events / %.1f MeV", 1e3 * s12_dat_hist.GetBinWidth(1)));

    TH1F s12_pdf_hist("s12_pdf_hist", "", s12.getNumBins(), s12.getLowerLimit(), s12.getUpperLimit());

    TH1F s13_dat_hist("s13_dat_hist", "", s13.getNumBins(), s13.getLowerLimit(), s13.getUpperLimit());
    s13_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV]");
    s13_dat_hist.GetYaxis()->SetTitle(TString::Format("Events / %.1f MeV", 1e3 * s13_dat_hist.GetBinWidth(1)));

    TH1F s13_pdf_hist("s13_pdf_hist", "", s13.getNumBins(), s13.getLowerLimit(), s13.getUpperLimit());

    TH1F s23_dat_hist("s23_dat_hist", "", s13.getNumBins(), s13.getLowerLimit(), s13.getUpperLimit());
    s23_dat_hist.GetXaxis()->SetTitle("m^{2}(#pi^{+} #pi^{-}) [GeV]");
    s23_dat_hist.GetYaxis()->SetTitle(TString::Format("Events / %.1f MeV", 1e3 * s13_dat_hist.GetBinWidth(1)));

    TH1F s23_pdf_hist("s23_pdf_hist", "", s13.getNumBins(), s13.getLowerLimit(), s13.getUpperLimit());

    double totalPdf = 0;
    double totalDat = 0;
    TH2F dalitzpp0_dat_hist("dalitzpp0_dat_hist",
                            "",
                            s12.getNumBins(),
                            s12.getLowerLimit(),
                            s12.getUpperLimit(),
                            s13.getNumBins(),
                            s13.getLowerLimit(),
                            s13.getUpperLimit());
    dalitzpp0_dat_hist.SetStats(false);
    dalitzpp0_dat_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV]");
    dalitzpp0_dat_hist.GetYaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
    TH2F dalitzpp0_pdf_hist("dalitzpp0_pdf_hist",
                            "",
                            s12.getNumBins(),
                            s12.getLowerLimit(),
                            s12.getUpperLimit(),
                            s13.getNumBins(),
                            s13.getLowerLimit(),
                            s13.getUpperLimit());

    dalitzpp0_pdf_hist.GetXaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
    dalitzpp0_pdf_hist.GetYaxis()->SetTitle("m^{2}(K^{-} K^{+}) [GeV^{2}]");
    dalitzpp0_pdf_hist.SetStats(false);
    std::vector<Observable> vars;
    vars.push_back(s12);
    vars.push_back(s13);
    vars.push_back(eventNumber);
    UnbinnedDataSet currData(vars);
    int evtCounter = 0;

    for(int i = 0; i < s12.getNumBins(); ++i) {
        s12.setValue(s12.getLowerLimit() + (s12.getUpperLimit() - s12.getLowerLimit()) * (i + 0.5) / s12.getNumBins());
        for(int j = 0; j < s13.getNumBins(); ++j) {
            s13.setValue(s13.getLowerLimit()
                         + (s13.getUpperLimit() - s13.getLowerLimit()) * (j + 0.5) / s13.getNumBins());
            if(!inDalitz(s12.getValue(), s13.getValue(), D_MASS, d1_MASS, d2_MASS, d3_MASS))
                continue;
            eventNumber.setValue(evtCounter);
            evtCounter++;
            currData.addEvent();
        }
    }
    overallSignal->setData(&currData);
    signaldalitz->setDataSize(currData.getNumEvents());
    std::vector<std::vector<double>> pdfValues = overallSignal->getCompProbsAtDataPoints();
    for(unsigned int j = 0; j < pdfValues[0].size(); ++j) {
        double currs12 = currData.getValue(s12, j);
        double currs13 = currData.getValue(s13, j);

        dalitzpp0_pdf_hist.Fill(currs12, currs13, pdfValues[0][j]);
        s12_pdf_hist.Fill(currs12, pdfValues[0][j]);
        s13_pdf_hist.Fill(currs13, pdfValues[0][j]);
        s23_pdf_hist.Fill(cpuGetM23(massSum,currs12, currs13), pdfValues[0][j]);
        totalPdf += pdfValues[0][j];
    }

    TCanvas foo;
    foo.SetLogz(false);
    dalitzpp0_pdf_hist.Draw("colz");

    foo.SaveAs("plots/dalitzpp0_pdf.png");

    for(unsigned int evt = 0; evt < Data->getNumEvents(); ++evt) {
        double data_s12 = Data->getValue(s12, evt);
        s12_dat_hist.Fill(data_s12);
        double data_s13 = Data->getValue(s13, evt);
        s13_dat_hist.Fill(data_s13);
        dalitzpp0_dat_hist.Fill(data_s12, data_s13);
        s23_dat_hist.Fill(cpuGetM23(massSum,data_s12, data_s13));
        totalDat++;
    }
    dalitzpp0_dat_hist.Draw("colz");
    foo.SaveAs("plots/dalitzpp0_dat.png");

    drawFitPlotsWithPulls(&s12_dat_hist, &s12_pdf_hist, plotdir);
    drawFitPlotsWithPulls(&s13_dat_hist, &s13_pdf_hist, plotdir);
    drawFitPlotsWithPulls(&s23_dat_hist, &s23_pdf_hist, plotdir);
}

void runtoyfit(std::string name){

    s12.setNumBins(1500);
    s13.setNumBins(1500);

    gettoydata(name);

    GOOFIT_INFO("Number of Events in dataset: {}", Data->getNumEvents());

    if(signaldalitz == nullptr){
    signaldalitz = makesignalpdf(0);
    }

    comps.clear();
    comps.push_back(signaldalitz);
    ProdPdf *overallsignal = new ProdPdf("overallsignal",comps);
    overallsignal->setData(Data);
    signaldalitz->setDataSize(Data->getNumEvents());

    FitManagerMinuit2 fitter(overallsignal);
    fitter.setVerbosity(3);

    auto func_min = fitter.fit();

    auto ff = signaldalitz->fit_fractions();

    PrintFF(ff);

    makeToyDalitzPdfPlots(overallsignal);

    //makeplot(signaldalitz->getDecayInfo().resonances.at(0),0);

}

int main(int argc, char **argv){

    GooFit::Application app{"D2PPP",argc,argv};

    size_t  nevents = 100000;
    auto gen = app.add_subcommand("gen","generate toy data");
    gen->add_option("-e,--events",nevents,"The number of events to generate",true);

    auto fit = app.add_subcommand("fit","fit toy data");


    GOOFIT_PARSE(app);

    /// Make the plot directory if it does not exist
    std::string command = "mkdir -p plots";
    if(system(command.c_str()) != 0)
        throw GooFit::GeneralError("Making `plots` directory failed");

    if(*gen){
        CLI::AutoTimer timer("MC Generation");
        runtoygen("D2PPP_toy.txt",nevents);
    }

    if(*fit){
        CLI::AutoTimer timer("FIT");
        runtoyfit("D2PPP_toy.txt");
    }

}
